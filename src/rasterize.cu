#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & Sarah Forcier
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>

// shading
#define BLINN 0 // exponent
#define NORMAL 0
#define POSITION 0
#define DEPTH 0
#define TEXTURE 1
#define BILINEAR 1
#define CORRECTED_PERSPECTIVE_TEXTURE 1

#define AMBIENT 1 // intensity percentage

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 screen_pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 pos;	// eye space position used for shading
		glm::vec3 nor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		glm::vec3 dcol;
		glm::vec3 scol;
		glm::vec2 texcoord0;
		TextureData* dev_diffuseTex = NULL;
		int texWidth, texHeight;
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 dColor;
		glm::vec3 sColor;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 pos;	// eye space position used for shading
		glm::vec3 nor;
		float depth;

		 VertexAttributeTexcoord texcoord0;
		 TextureData* dev_diffuseTex;
		 int texWidth, texHeight;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static int * dev_depth = NULL;	// you might need this buffer when doing depth test
static int * dev_depthMutex = NULL;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) 
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

__host__ __device__ glm::vec3 bilinearFilter(float u, float v, int width, TextureData* texture)
{
	float x = glm::floor(u);
	float y = glm::floor(v);
	float dx = u - x;
	float dy = v - y;

	int offset = 3 * ((int)x + (int)y * width);
	glm::vec3 c00 = glm::vec3(texture[offset] / 255.f, texture[offset + 1] / 255.f, texture[offset + 2] / 255.f);
	glm::vec3 c01 = glm::vec3(texture[offset + 3] / 255.f, texture[offset + 4] / 255.f, texture[offset + 5] / 255.f);
	offset = 3 * ((int)x + (int)(y + 1) * width);
	glm::vec3 c10 = glm::vec3(texture[offset] / 255.f, texture[offset + 1] / 255.f, texture[offset + 2] / 255.f);
	glm::vec3 c11 = glm::vec3(texture[offset + 3] / 255.f, texture[offset + 4] / 255.f, texture[offset + 5] / 255.f);

	glm::vec3 mix0 = glm::mix(c00, c01, dx);
	glm::vec3 mix1 = glm::mix(c10, c11, dx);

	return glm::mix(mix0, mix1, dy);
}

__host__ __device__ glm::vec3 shade(Fragment frag)
{
    glm::vec3 finalColor = glm::vec3(0.f);

	// TODO: add your fragment shader code here
	glm::vec3 lightPosition = glm::vec3(0.f, 0.f, 0.f); // hard coded light
	glm::vec3 normal = frag.nor;
    glm::vec3 lightDir = glm::normalize(lightPosition - frag.pos);
    finalColor = glm::dot(normal, lightDir) * frag.dColor;

#if TEXTURE
	if (frag.dev_diffuseTex != NULL) {
		float u = frag.texcoord0.x * frag.texWidth;
		float v = frag.texcoord0.y * frag.texHeight;
		
#if BILINEAR
		finalColor *= bilinearFilter(u, v, frag.texWidth, frag.dev_diffuseTex);
#else 
		int offset = 3 * ((int)u + (int)v * frag.texWidth);
		finalColor *= glm::vec3(frag.dev_diffuseTex[offset + 0] / 255.f,
			frag.dev_diffuseTex[offset + 1] / 255.f,
			frag.dev_diffuseTex[offset + 2] / 255.f);
#endif
	}
#endif
#if AMBIENT
	finalColor += (float)AMBIENT / 100.f * glm::vec3(1.f);
#endif

#if BLINN 
    glm::vec3 viewDir = - frag.pos; // camera is at origin in eye space
    glm::vec3 half = glm::normalize(lightDir + viewDir);
    finalColor += glm::pow(glm::dot(normal, half), BLINN) * frag.sColor;
#endif
  
// debugging
#if NORMAL 
    finalColor = frag.nor;

#elif POSITION
    finalColor = frag.pos;

#elif DEPTH
    finalColor = glm::vec3(frag.depth);

#endif

    return finalColor;
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
    	framebuffer[index] = shade(fragmentBuffer[index]);
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) 
{
    width = w;
    height = h;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

	hipFree(dev_depthMutex);
	hipMalloc(&dev_depthMutex, sizeof(int));
	hipMemset(dev_depthMutex, 0, sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) 
{
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) 
{

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) 
{
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) 
{

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}
}

__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) 
{

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid >= numVertices) return;

	// TODO: Apply vertex transformation here
	// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
	glm::vec4 world_pos = glm::vec4(primitive.dev_position[vid], 1.f);
	glm::vec4 clip_pos = MVP * world_pos;

	// Finally transform x and y to viewport space
	primitive.dev_verticesOut[vid].screen_pos.x = 0.5f * (float)width * (clip_pos.x / clip_pos.w + 1.f);
	primitive.dev_verticesOut[vid].screen_pos.y = 0.5f * (float)height * (1.f - clip_pos.y / clip_pos.w);
	primitive.dev_verticesOut[vid].screen_pos.z = -0.5f * (clip_pos.z / clip_pos.w + 1.f);

	// TODO: Apply vertex assembly here
	// Assemble all attribute arrays into the primitive array
	primitive.dev_verticesOut[vid].pos = glm::vec3(MV * world_pos);
	primitive.dev_verticesOut[vid].nor = glm::normalize(MV_normal * primitive.dev_normal[vid]);
	primitive.dev_verticesOut[vid].dcol = glm::vec3(1.f);
	primitive.dev_verticesOut[vid].scol = glm::vec3(1.f);
#if TEXTURE
	primitive.dev_verticesOut[vid].texWidth = primitive.diffuseTexWidth;
	primitive.dev_verticesOut[vid].texHeight = primitive.diffuseTexHeight;
	primitive.dev_verticesOut[vid].dev_diffuseTex = primitive.dev_diffuseTex;
	primitive.dev_verticesOut[vid].texcoord0 = primitive.dev_texcoord0[vid];
#endif
}

static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) 
{
	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}

		// TODO: other primitive types (point, line)
	}	
}

__global__
void _kernRasterize(int numPrimitives, int width, Primitive *primitives, Fragment *fragmentBuffer, int *depthBuffer, int *depthMutex)
{
	// primitive id
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id >= numPrimitives) return;

	Primitive prim = primitives[id];
	glm::vec3 vec_arr[3];
	vec_arr[0] = glm::vec3(prim.v[0].screen_pos);
	vec_arr[1] = glm::vec3(prim.v[1].screen_pos);
	vec_arr[2] = glm::vec3(prim.v[2].screen_pos);

	AABB bbox = getAABBForTriangle(vec_arr);

	for (int i = bbox.min.x; i < bbox.max.x; ++i) {
		for (int j = bbox.min.y; j < bbox.max.y; ++j) {
			
			glm::vec3 bary = calculateBarycentricCoordinate(vec_arr, glm::vec2(i, j));
			if (isBarycentricCoordInBounds(bary)) {

				int index = i + (j * width);

				float d = getZAtCoordinate(bary, vec_arr);
				int depth = d * INT_MAX;
				bool isSet;
				bool isMin = false;
				do {
					isSet = (atomicCAS(depthMutex, 0, 1) == 0);
					if (isSet) {
						if (depth < depthBuffer[index]) {
							depthBuffer[index] = depth;
							isMin = true;
						}

						*depthMutex = 0;
					}
				} while (!isSet);

				//atomicMin(&depthBuffer[index], depth);
					
				//if (depthBuffer[index] == depth) {
				if (isMin) {
					fragmentBuffer[index].dColor = bary.x * prim.v[0].dcol + 
												   bary.y * prim.v[1].dcol + 
												   bary.z * prim.v[2].dcol;

#if TEXTURE
					
#if CORRECTED_PERSPECTIVE_TEXTURE
					float w0 = 1.f / prim.v[0].pos.z;
					float w1 = 1.f / prim.v[1].pos.z;
					float w2 = 1.f / prim.v[2].pos.z;

					fragmentBuffer[index].texcoord0 = bary.x * prim.v[0].texcoord0 * w0 +
						                              bary.y * prim.v[1].texcoord0 * w1 +
						                              bary.z * prim.v[2].texcoord0 * w2;
					fragmentBuffer[index].texcoord0 /= (bary.x * w0 + bary.y * w1 + bary.z * w2);

#else 
					fragmentBuffer[index].texcoord0 = bary.x * prim.v[0].texcoord0 +
						                              bary.y * prim.v[1].texcoord0 +
						                              bary.z * prim.v[2].texcoord0;
#endif

					fragmentBuffer[index].dev_diffuseTex = prim.v[0].dev_diffuseTex;
					fragmentBuffer[index].texWidth = prim.v[0].texWidth;
					fragmentBuffer[index].texHeight = prim.v[0].texHeight;
#endif

					fragmentBuffer[index].sColor = bary.x * prim.v[0].scol + 
												   bary.y * prim.v[1].scol + 
												   bary.z * prim.v[2].scol;

					fragmentBuffer[index].nor = bary.x * prim.v[0].nor + 
												bary.y * prim.v[1].nor + 
												bary.z * prim.v[2].nor;

					fragmentBuffer[index].pos = bary.x * prim.v[0].pos + 
												bary.y * prim.v[1].pos + 
												bary.z * prim.v[2].pos;

					fragmentBuffer[index].depth = d;
				}
			}
		}
	}

}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) 
{
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);
	dim3 numThreadsPerBlock(128);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	
	// TODO: rasterize
	dim3 numBlocksForPrimitives((totalNumPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
	_kernRasterize << <numBlocksForPrimitives, numThreadsPerBlock >> > (totalNumPrimitives, width, dev_primitives, dev_fragmentBuffer, dev_depth, dev_depthMutex);

    // Copy depthbuffer colors into framebuffer
	render << <blockCount2d, blockSize2d >> >(width, height, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() 
{
    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

	hipFree(dev_depthMutex);
	dev_depthMutex = NULL;

    checkCUDAError("rasterize Free");
}
